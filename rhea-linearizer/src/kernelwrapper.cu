#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "math.h"

typedef
union
{
  int32_t i;
  struct
  {
    int16_t lo; // endian-specific!
    int16_t hi;
  };
} fixed_point;

void checkCUDAError(const char* msg);

__global__ void kernel(char *pairpixelsD, int nP, int width, unsigned char *pairsD,
    float* outmD, int degree, const int xheight, int nPairs) {

  // just use global memory for now
  // get threadID:
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= nPairs) return;
  // first, get the first and second pixel from pairsD
  unsigned char *pairPtr = &pairsD[6*idx];
  int x0 = ( pairPtr[0] << 8 ) + pairPtr[1];
  int x1 = ( pairPtr[2] << 8 ) + pairPtr[3];
  int y0 = pairPtr[4];
  int y1 = pairPtr[5];
 
  // calculate the first three variables
  float vdx = (x1 - x0)/(float)xheight;
  float vdy = (y1 - y0)/(float)xheight;
  float vyy = 0.5*(y0 + y1)/(float)xheight;

  // now calculate amount of black
  int btotal = 0;
  int bblack = 0;
  fixed_point f;

  if(abs(y1-y0) < abs(x1-x0)) {
    int x;
    int32_t m=((int32_t)(y1-y0)<<16)/(x1-x0);

    f.i=y0<<16;
    for (x=x0;x<=x1;x++,f.i+=m)
    {
      fixed_point g=f;
      g.i+=32767;
      btotal++;

      bblack += pairpixelsD[width * g.hi + x];
    }
  } else {
    int y;
    int32_t m=((int32_t)(x1-x0)<<16)/(y1-y0);

    f.i=x0<<16;
    for (y=y0;y<=y1;y++,f.i+=m)
    {
      fixed_point g=f;
      g.i+=32767;
      btotal++;
      bblack += pairpixelsD[width * y + g.hi];
    }
  }

  float vbl = bblack/(float)btotal;

  // now calculate monomial results and store
  // should maybe be done locally and then pushed to global memory
  for(int evdx = 0; evdx <= degree; evdx++) {
    for(int evdy = 0; evdy <= degree; evdy++) {
      for(int evyy = 0; evyy <= degree; evyy++) {
        for(int evbl = 0; evbl <= degree; evbl++) {
          outmD[evbl + degree*evyy + (degree*degree)*evdy +
            (degree*degree*degree)*evdx] += powf(vdx, evdx) + powf(vdy, evdy)
            + powf(vyy, evyy) + powf(vbl, evbl);
        }
      }
    }
  }
}

void kernel_wrapper(char* pairpixelsH, int width, int height, unsigned char*
    pairsH, int nPairs, float* outmH, const int degree, const int nMonomials,
    const int xheight) {

  // create matrix with both letters in it
  // one-dimensional, with one byte per pixel, going from bottom to top,
  // left to right.
  // also, create matrix for polynomial output on device
  float *outmD;
  hipMalloc((void**) &outmD, nMonomials*sizeof(float));
  hipMemset(outmD, 0.f, nMonomials*sizeof(float));

  // copy matrix into CUDA memory
  char *pairpixelsD;
  hipMalloc((void**) &pairpixelsD, width*height*sizeof(char));
  hipMemcpy(pairpixelsD, pairpixelsH, width*height*sizeof(char), hipMemcpyHostToDevice);

  // copy list of pixel pairs into CUDA memory
  unsigned char* pairsD;
  hipMalloc((void**) &pairsD, nPairs*6*sizeof(unsigned char));
  
  checkCUDAError("Couldn't create pairs");
  hipMemcpy(pairsD, pairsH, nPairs*6*sizeof(unsigned char), hipMemcpyHostToDevice);
  checkCUDAError("Couldn't copy pairs");
  // create tons of threads with ID composed of left and right
  int nThreadsPerBlock = 512;
  int nBlocks = ceil(nPairs/(float)nThreadsPerBlock);

  // calculate number of shared memory bytes needed
  printf("xheight: %d\n", xheight);
  // call the kernel
  kernel<<< nBlocks, nThreadsPerBlock >>>( pairpixelsD, width*height, width, pairsD, outmD, degree, xheight, nPairs);
  hipDeviceSynchronize();
  checkCUDAError("Kernel Failed!");

  // copy over results
  hipMemcpy(outmH, outmD, nMonomials*sizeof(float), hipMemcpyDeviceToHost);

  // finally: free memory
  hipFree(pairpixelsD);
  hipFree(pairsD);
  hipFree(outmD);

}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err)
  {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }
}
